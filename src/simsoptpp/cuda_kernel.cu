#include "hip/hip_runtime.h"
// #include "simdhelpers.h" // import above cuda_runtime to prevent collision for rsqrt
#include <hip/hip_runtime.h>
#include <iostream>
#include "tracing.h"
#include <math.h>
#include "xtensor-python/pyarray.hpp"     // Numpy bindings
typedef xt::pyarray<double> PyArray;
#include "xtensor-python/pytensor.hpp"     // Numpy bindings
typedef xt::pytensor<double, 2, xt::layout_type::row_major> PyTensor;
using std::shared_ptr;
using std::vector;
namespace py = pybind11;
#include <fmt/core.h>

// #include <Eigen/Core>

#include "magneticfield.h"
#include "boozermagneticfield.h"
#include "regular_grid_interpolant_3d.h"

// #define dt 1e-7

// Particle Data Structure
typedef struct particle_t {
    double x;  // Position X
    double y;  // Position Y
    double z;  // Position Z
    double v_par; // Velocity parallel
    double v_perp; // Velocity perpendicular
    double dotx;
    double doty;
    double dotz;
    double dotv_par;
    bool has_left;
} particle_t;

typedef struct workspace_t {
    double r_shape[4];
    double phi_shape[4];
    double z_shape[4];

    double r_dshape[4];
    double phi_dshape[4];
    double z_dshape[4];

    double B[3];
    double grad_B[9];
    double nabla_normB[3];
    double cross_prod[3];
} workspace_t;

__global__ void addKernel(int *c, const int* a, const int* b, int size){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx < size){
        c[idx] = a[idx] + b[idx];
    }
}

extern "C" void addKernelWrapper(int *c, const int *a, const int *b, int size){
    int *d_a, *d_b, *d_c;

    hipMalloc((void **)&d_a, size*sizeof(int));
    hipMalloc((void **)&d_b, size*sizeof(int));
    hipMalloc((void **)&d_c, size*sizeof(int));

    hipMemcpy(d_a, a, size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size*sizeof(int), hipMemcpyHostToDevice);

    addKernel<<<1, 256>>>(d_c, d_a, d_b, size);

    for(int i=0; i<size; ++i){
        // // std::cout << c[i] <<"\n";
    }

    hipMemcpy(c, d_c, size*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}





__host__ __device__ void shape(double x, double* shape){
    shape[0] = (1.0-x)*(2.0-x)*(3.0-x)/6.0;
    shape[1] = x*(2.0-x)*(3.0-x)/2.0;
    shape[2] = x*(x-1.0)*(3.0-x)/2.0;
    shape[3] = x*(x-1.0)*(x-2.0)/6.0;
    return;         
}

__host__ __device__ void dshape(double x, double h, double* dshape){
    dshape[0] = (-(2.0-x)*(3.0-x)-(1.0-x)*(3.0-x)-(1.0-x)*(2.0-x))/(h*6.0);
    dshape[1] = ( (2.0-x)*(3.0-x)-x*(3.0-x)-x*(2.0-x))/(h*2.0);
    dshape[2] = ( (x-1.0)*(3.0-x)+x*(3.0-x)-x*(x-1.0))/(h*2.0);
    dshape[3] = ( (x-1.0)*(x-2.0)+x*(x-2.0)+x*(x-1.0))/(h*6.0);
    return;         
}

// state has 5 elements: current values of x,y,z,v_perp
// // compute derivs for: x, y, z, v_perp
// void calc_derivs(double* state, double* derivs, double* rrange_arr, double* zrange_arr, double* phirange_arr, double* quadpts_arr,
//                         double dt, double tmax, double m, double q) {
//     double x = state[0];
//     double y = state[1];
//     double z = state[2];
//     double v_perp = state[3];

//     double r_shape[4];
//     double phi_shape[4];
//     double z_shape[4];

//     double r_dshape[4];
//     double phi_dshape[4];
//     double z_dshape[4];

//     double B[3];
//     double grad_B[9];
//     double nabla_normB[3];
//     double cross_prod[3];

//     double r_grid_size = (rrange_arr[1] - rrange_arr[0]) / (rrange_arr[2]-1);
//     double phi_grid_size = 2*M_PI / phirange_arr[2];
//     double z_grid_size = (zrange_arr[1] - zrange_arr[0]) / (zrange_arr[2]-1);

//     // interpolate B field for current state
//     double r = sqrt(x*x + y*y);
//     double phi = atan2(y, x);

//     // index into mesh to obtain nearby points
//     int i = (int) ((r - rrange_arr[0]) / r_grid_size) + 1;
//     int j = (int) ((z - zrange_arr[0]) / z_grid_size) + 1;
//     int k = (int) (phi / phi_grid_size) + 1;



//     // normalized positions in local grid wrt e.g. r at index i
//     int nr = rrange_arr[2];
//     int nphi = phirange_arr[2];
//     int nz = zrange_arr[2];
//     double r_rel = (r -  (rrange_arr[0] + i*r_grid_size)) / r_grid_size;
//     double z_rel = (z -  (zrange_arr[0] + j*z_grid_size)) / z_grid_size;
//     double phi_rel = (phi - (k*phi_grid_size)) / phi_grid_size;


//     // std::cout << r << "\t" << -1*(r_rel*r_grid_size - r) << "\t" << r_grid_size << "\n";
//     // std::cout << z << "\t" << -1*(z_rel*z_grid_size - z) << "\t" << z_grid_size << "\n";
//     // std::cout << phi << "\t" << -1*(phi_rel*phi_grid_size - phi) << "\t" << phi_grid_size << "\n";
//     // std::cout << "using index " <<  (i*nz*nphi + j*nphi + k) << "\n";
//     // std::cout << quadpts_arr[4*(i*nz*nphi + j*nphi + k) + 3] << "\n";
//     // // std::cout << "grid point found \n";

//     // // std::cout << "r_rel " << r_rel << "\t" << z_rel << "\t" << phi_rel << "\n";

//     shape(r_rel, r_shape);
//     shape(z_rel, z_shape);
//     shape(phi_rel, phi_shape);


//     // // std::cout <<"shape set \n";
//     // accumulate interpolation of B
//     B[0] = 0.0;
//     B[1] = 0.0;            
//     B[2] = 0.0;

//     // interpolate the distance to the surface
//     double surface_dist = 0.0;

//     // // std::cout << "starting B accumulation\n";
//     // quad pts are indexed r z phi
//     bool is_lost = false;
//     for(int ii=0; ii<=3; ++ii){             
//         for(int jj=0; jj<=3; ++jj){                 
//             for(int kk=0; kk<=3; ++kk){
//                 int wrap_k = ((k+kk-1) % nphi) + 1;

//                 if ((i+ii >= 0 & i+ii < nr) & (j+jj >= 0 & j+jj < nz)){
//                     int start = 4*((i+ii)*nz*nphi + (j+jj)*nphi + (wrap_k));
//                     // // std::cout << "start=" << start << "\t" << 4*nr*nz*nphi << "\n";
//                     B[0] += quadpts_arr[start]   * r_shape[ii]*z_shape[jj]*phi_shape[kk];
//                     B[1] += quadpts_arr[start+1] * r_shape[ii]*z_shape[jj]*phi_shape[kk];
//                     B[2] += quadpts_arr[start+2] * r_shape[ii]*z_shape[jj]*phi_shape[kk];

//                     is_lost = is_lost || (quadpts_arr[start+3] < 0); 
//                     // // std::cout << ii << "\t" << jj << "\t" << kk << "\n";
//                     // // std::cout << "interp surface dist val: " << quadpts_arr[start+3] << "\n";
//                     surface_dist += quadpts_arr[start+3] * r_shape[ii]*z_shape[jj]*phi_shape[kk];
//                 } else{
//                     // // std::cout << "bad grid index for" << r << "\t" << phi << "\t" << z <<"\n"; 
//                 }

//             }
//         }
//     }

// }

// out contains derivatives for x , y, z, v_par, and then norm of B and surface distance interpolation
__host__ __device__ void calc_derivs(double* state, double* out, double* rrange_arr, double* zrange_arr, double* phirange_arr, double* quadpts_arr, double m, double q, double mu){
    double r_shape[4];
    double phi_shape[4];
    double z_shape[4];

    double r_dshape[4];
    double phi_dshape[4];
    double z_dshape[4];

    double B[3];
    double grad_B[9];
    double nabla_normB[3];
    double cross_prod[3];

    double r_grid_size = (rrange_arr[1] - rrange_arr[0]) / (rrange_arr[2]-1);
    double phi_grid_size = 2*M_PI / (phirange_arr[2] - 1);
    double z_grid_size = (zrange_arr[1] - zrange_arr[0]) / (zrange_arr[2]-1);
    

    double x = state[0];
    double y = state[1];
    double z = state[2];
    double v_par = state[3];
    // double v_perp = state[4];

    // std::cout << "load v_par " << v_par;

    // magnetic field quad points are in cylindrical coordinates
    double r = sqrt(x*x + y*y);
    double phi = atan2(y, x); 
    
    // keep phi positive
    phi += (2*M_PI)*(phi < 0);
    
    // fmt::print("r z phi: {} {} {}\n", r, z, phi);
    // std::cout << std::format("r z phi: {} {} {}\n", r, z , phi);
    // std::cout << "x y z " << x << "\t" << y << "\t" << z << "\n";


    // index into mesh to obtain nearby points
    // get correct "meta grid" for continuity
    int i = 3*((int) ((r - rrange_arr[0]) / r_grid_size) / 3);
    int j = 3*((int) ((z - zrange_arr[0]) / z_grid_size) / 3);
    int k = 3*((int) (phi / phi_grid_size) / 3);
    // int k = 3*((int) ((phi+M_PI) / phi_grid_size) / 3);


    // std::cout << "i j k " <<  i << "\t" << j << "\t" << k << "\n"; 
    // std::cout << "phi_grid_size: " << phi_grid_size << "\n"; 

    // std::cout << "indices: " << i << "\t" << r << "\t" << rrange_arr[0] << "\t" << r_grid_size << "\n";
    // std::cout << "position: " << x << "\t" << y << "\t" << z <<"\n";

    // normalized positions in local grid wrt e.g. r at index i
    int nr = rrange_arr[2];
    int nphi = phirange_arr[2];
    int nz = zrange_arr[2];
    double r_rel = (r -  (rrange_arr[0] + i*r_grid_size)) / r_grid_size;
    double z_rel = (z -  (zrange_arr[0] + j*z_grid_size)) / z_grid_size;
    // double phi_rel = M_PI*(2*(k % nphi) - nphi) / phi_grid_size;
    double phi_rel = (phi - (k*phi_grid_size)) / phi_grid_size;
    // fmt::print("r_rel z_rel phi_rel: {} {} {}\n", r_rel, z_rel, phi_rel);


    // std::cout << r << "\t" << -1*(r_rel*r_grid_size - r) << "\t" << r_grid_size << "\n";
    // std::cout << z << "\t" << -1*(z_rel*z_grid_size - z) << "\t" << z_grid_size << "\n";
    // std::cout << phi << "\t" << -1*(phi_rel*phi_grid_size - phi) << "\t" << phi_grid_size << "\n";
    // std::cout << i << "\t" << j << "\t" << k << "\n";
    // std::cout << "using index " <<  (i*nz*nphi + j*nphi + k) << "\n";
    // std::cout << quadpts_arr[4*(i*nz*nphi + j*nphi + k) + 3] << "\n";
    // // std::cout << "grid point found \n";

    // // std::cout << "r_rel " << r_rel << "\t" << z_rel << "\t" << phi_rel << "\n";

    shape(r_rel, r_shape);
    shape(z_rel, z_shape);
    shape(phi_rel, phi_shape);


    // // std::cout <<"shape set \n";
    // accumulate interpolation of B
    B[0] = 0.0;
    B[1] = 0.0;            
    B[2] = 0.0;

    // interpolate the distance to the surface
    double surface_dist = 0.0;

    // // std::cout << "starting B accumulation\n";
    // quad pts are indexed r z phi
    bool is_lost = false;
    for(int ii=0; ii<=3; ++ii){             
        for(int jj=0; jj<=3; ++jj){                 
            for(int kk=0; kk<=3; ++kk){
                int wrap_k = ((k+kk) % nphi);
                if ((i+ii >= 0 & i+ii < nr) & (j+jj >= 0 & j+jj < nz)){
                    // fmt::print("indices: {} {} {}\n", i+ii, j+jj, k+kk);
                    int start = 4*((i+ii)*nz*nphi + (j+jj)*nphi + (wrap_k));
                    // // std::cout << "start=" << start << "\t" << 4*nr*nz*nphi << "\n";
                    B[0] += quadpts_arr[start]   * r_shape[ii]*z_shape[jj]*phi_shape[kk];
                    B[1] += quadpts_arr[start+1] * r_shape[ii]*z_shape[jj]*phi_shape[kk];
                    B[2] += quadpts_arr[start+2] * r_shape[ii]*z_shape[jj]*phi_shape[kk];

                    is_lost = is_lost || (quadpts_arr[start+3] < 0); 
                    // // std::cout << ii << "\t" << jj << "\t" << kk << "\n";
                    // // std::cout << "interp surface dist val: " << quadpts_arr[start+3] << "\n";
                    surface_dist += quadpts_arr[start+3] * r_shape[ii]*z_shape[jj]*phi_shape[kk];
                } else{
                    // // std::cout << "bad grid index for" << r << "\t" << phi << "\t" << z <<"\n"; 
                }

            }
        }
    }

    // std::cout << "k " << k << "\t" << nphi << "\n";


    // std::cout << "is quad pt lost: " << is_lost << "\n";
    if(!is_lost){ // can't lose a particle if no quad pts are lost
        surface_dist = 1.0;    
    }
    // // std::cout << "B interpolated \n";

    // // std::cout << "r=" << r << "\t" << x << "\t" << y << "\t" << p.v_par << "\t" << surface_dist << "\n";

    // // std::cout << "particle not lost \n";

    //  Interpolate grad B: columns are partial deriv wrt r, z, phi, rows are entries of B
    //  row major order
    for(int ii=0; ii<9; ++ii){
        grad_B[ii] = 0.0;
    }
    dshape(r_rel, r_grid_size, r_dshape);
    dshape(phi_rel, phi_grid_size, phi_dshape);
    dshape(z_rel, z_grid_size, z_dshape);

    for(int ii=0; ii<=3; ++ii){             
        for(int jj=0; jj<=3; ++jj){                 
            for(int kk=0; kk<=3; ++kk){
                int wrap_k = ((k+kk) % nphi);
                if ((i+ii >= 0 & i+ii < nr) & (j+jj >= 0 & j+jj < nz)){
                    int start = 4*((i+ii)*nz*nphi + (j+jj)*nphi + (wrap_k));
                    // interpolate gradient for each entry of B, filling in each row of the gradient
                    for(int l=0; l<3; ++l){
                        double Bval = quadpts_arr[start+l];
                        grad_B[3*l]   += Bval * r_dshape[ii]*z_shape[jj]*phi_shape[kk];
                        grad_B[3*l+1] += Bval * r_shape[ii]*z_dshape[jj]*phi_shape[kk];
                        grad_B[3*l+2] += Bval * r_shape[ii]*z_shape[jj]*phi_dshape[kk];
                    }
                }

            }
        }
    }

    // // std::cout << "grad B interpolated \n";


    // convert gradient from cylindrical (r, z, phi) to cartesian coordinates (x, y, z)
    double c = cos(phi);
    double s = sin(phi);


    for(int l=0; l<3; ++l){ // iter over row
        double dfdr = grad_B[3*l];
        double dfdphi_divr = grad_B[3*l+2] / r;
        
        grad_B[3*l]   = c*dfdr - s*dfdphi_divr;
        grad_B[3*l+2] = grad_B[3*l+1]; // z index changes
        grad_B[3*l+1] = s*dfdr + c*dfdphi_divr;
    }

    // fmt::print("B: {} {} {}\n", B[0], B[1], B[2]);
    // std::cout << "B " << B[0] << "\t" << B[1] << "\t" << B[2] << "\n";
    // return;
    // std::cout << "grad_B" << grad_B[0] << "\t" << grad_B[1] << "\t" << grad_B[2] << "\n";
    // now compute derivatives

    // // std::cout << "starting updates \n";

    double normB = sqrt(B[0]*B[0] + B[1]*B[1] + B[2]*B[2]);


    // compute \nabla |B|
    //  \nabla |B| = (\nabla B  B) / (2 |B|)
    nabla_normB[0] = (grad_B[0]*B[0] + grad_B[1]*B[1] + grad_B[2]*B[2]) / (normB);
    nabla_normB[1] = (grad_B[3]*B[0] + grad_B[4]*B[1] + grad_B[5]*B[2]) / (normB);
    nabla_normB[2] = (grad_B[6]*B[0] + grad_B[7]*B[1] + grad_B[8]*B[2]) / (normB);

    // compute B \times \nabla |B|
    cross_prod[0] = B[1]*nabla_normB[2] - B[2]*nabla_normB[1];
    cross_prod[1] = B[2]*nabla_normB[0] - B[0]*nabla_normB[2];
    cross_prod[2] = B[0]*nabla_normB[1] - B[1]*nabla_normB[0];

    // std::cout << "compute x deriv: " << v_par << "\t" << B[0] << "\t" << normB << "\t" <<  v_par << "\t" << cross_prod[0] << "\t" << m << "\t" << q << "\n";

    double v_perp2 = 2*mu*normB;

    // std::cout << "should be 0: " << (0.5*v_perp2 + pow(v_par, 2))*cross_prod[0] * m/(q*pow(normB, 3)) << "\n";
    // std::cout << "should be positive " << v_par * B[0]/normB << "\n";
    // std::cout << "v_par" << v_par << "\n";

    out[0] = v_par * B[0]/normB + (0.5*v_perp2 + pow(v_par, 2))*cross_prod[0] * m/(q*pow(normB, 3));
    out[1] = v_par * B[1]/normB + (0.5*v_perp2 + pow(v_par, 2))*cross_prod[1] * m/(q*pow(normB, 3));
    out[2] = v_par * B[2]/normB + (0.5*v_perp2 + pow(v_par, 2))*cross_prod[2] * m/(q*pow(normB, 3));

    double BdotNablaNormB = B[0]*nabla_normB[0] + B[1]*nabla_normB[1] + B[2]*nabla_normB[2];
    out[3] = -mu*BdotNablaNormB/normB;
    out[4] = normB;
    out[5] = surface_dist;

}


__host__  void trace_particle(particle_t& p, double* rrange_arr, double* zrange_arr, double* phirange_arr, double* quadpts_arr,
                        double dt, double tmax, double m, double q){
    double mu;
    int nsteps = (int) (tmax / dt);
    double surface_dist;
    // // std::cout << tmax << "\t" << dt << "\t" << nsteps << "\n";
    // double r_shape[4];
    // double phi_shape[4];
    // double z_shape[4];

    // double r_dshape[4];
    // double phi_dshape[4];
    // double z_dshape[4];

    // double B[3];
    // double grad_B[9];
    // double nabla_normB[3];
    // double cross_prod[3];

    // double r_grid_size = (rrange_arr[1] - rrange_arr[0]) / (rrange_arr[2]-1);
    // double phi_grid_size = 2*M_PI / phirange_arr[2];
    // double z_grid_size = (zrange_arr[1] - zrange_arr[0]) / (zrange_arr[2]-1);
    

    double t = 0.0;

    double state[4];
    state[0] = p.x;
    state[1] = p.y;
    state[2] = p.z;
    state[3] = p.v_par;
    // state[4] = p.v_perp;

    double derivs[6];

    // dummy call to get norm B
    calc_derivs(state, derivs, rrange_arr, zrange_arr, phirange_arr, quadpts_arr, m, q, -1);
    mu = p.v_perp*p.v_perp/(2*derivs[4]);

    const double a21 = 1.0 / 5.0;
    const double a31 = 3.0 / 40.0, a32 = 9.0 / 40.0;
    const double a41 = 44.0 / 45.0, a42 = -56.0 / 15.0, a43 = 32.0 / 9.0;
    const double a51 = 19372.0 / 6561.0, a52 = -25360.0 / 2187.0, a53 = 64448.0 / 6561.0, a54 = -212.0 / 729.0;
    const double a61 = 9017.0 / 3168.0, a62 = -355.0 / 33.0, a63 = 46732.0 / 5247.0, a64 = 49.0 / 176.0, a65 = -5103.0 / 18656.0;
    const double b1 = 35.0 / 384.0, b3 = 500.0 / 1113.0, b4 = 125.0 / 192.0, b5 = -2187.0 / 6784.0, b6 = 11.0 / 84.0;
    // const double bhat1 = 5179.0 / 57600.0, bhat3 = 7571.0 / 16695.0, bhat4 = 393.0 / 640.0, bhat5 = -92097.0 / 339200.0, bhat6 = 187.0 / 2100.0, bhat7 = 1.0 / 40.0;
    const double bhat1 = 71.0 / 57600.0, bhat3 = -71.0 / 16695.0, bhat4 = 71.0 / 1920.0, bhat5 = -17253.0 / 339200.0, bhat6 = 22.0 / 525.0, bhat7 = -1.0 / 40.0;


    double k2[6], k3[6], k4[6], k5[6], k6[6], k7[6];
    double x_temp[4], x_new[4], x_err[4];


    int counter = 0;
    while(t < tmax){
        // if(counter % 10 == 0){
        //     std::cout << "position: " << p.x << "\t" << p.y << "\t" << p.z << "\t" << "t=" << t  << "\t dt= " << dt << "\n";
        // }
        counter++;
        // std::cout << "Time: " << t << "\n";
        /*
        * Time step ODE
        * runge-kutta 4 (see https://lpsa.swarthmore.edu/NumInt/NumIntFourth.html)
        * 
        * Adaptive Dopri5 time step: p.167
        * https://link.springer.com/book/10.1007/978-3-540-78862-1
        */

        // compute k1
        state[0] = p.x;
        state[1] = p.y;
        state[2] = p.z;
        state[3] = p.v_par;

        calc_derivs(state, derivs, rrange_arr, zrange_arr, phirange_arr, quadpts_arr, m, q, mu);
        // return;
        // stop if particle lost
        surface_dist = derivs[5];
        if(surface_dist <= 0){
            // std::cout << "particle lost: " << surface_dist << "\t" << t << "\t" << dt << "\n";
            p.has_left = true;
            return;
        }
        
        // Compute k2
        for (int i = 0; i < 4; i++) x_temp[i] = state[i] + dt * a21 * derivs[i];
        calc_derivs(x_temp, k2, rrange_arr, zrange_arr, phirange_arr, quadpts_arr, m, q, mu);

        // Compute k3
        for (int i = 0; i < 4; i++) x_temp[i] = state[i] + dt * (a31 * derivs[i] + a32 * k2[i]);
        calc_derivs(x_temp, k3, rrange_arr, zrange_arr, phirange_arr, quadpts_arr, m, q, mu);

        // Compute k4
        for (int i = 0; i < 4; i++) x_temp[i] = state[i] + dt * (a41 * derivs[i] + a42 * k2[i] + a43 * k3[i]);
        calc_derivs(x_temp, k4, rrange_arr, zrange_arr, phirange_arr, quadpts_arr, m, q, mu);

        // Compute k5
        for (int i = 0; i < 4; i++) x_temp[i] = state[i] + dt * (a51 * derivs[i] + a52 * k2[i] + a53 * k3[i] + a54 * k4[i]);
        calc_derivs(x_temp, k5, rrange_arr, zrange_arr, phirange_arr, quadpts_arr, m, q, mu);

        // Compute k6
        for (int i = 0; i < 4; i++) x_temp[i] = state[i] + dt * (a61 * derivs[i] + a62 * k2[i] + a63 * k3[i] + a64 * k4[i] + a65 * k5[i]);
        calc_derivs(x_temp, k6, rrange_arr, zrange_arr, phirange_arr, quadpts_arr, m, q, mu);

        // Compute new state
        for (int i = 0; i < 4; i++) {
            x_new[i] = state[i] + dt * (b1 * derivs[i] + b3 * k3[i] + b4 * k4[i] + b5 * k5[i] + b6 * k6[i]);
        }

        // Compute k7 for error estimation
        calc_derivs(x_new, k7, rrange_arr, zrange_arr, phirange_arr, quadpts_arr, m, q, mu);
        
        // Compute  error
        // https://live.boost.org/doc/libs/1_82_0/libs/numeric/odeint/doc/html/boost_numeric_odeint/odeint_in_detail/steppers.html
        double tol=1e-9;
        // std::cout << "error elts \n";
        double err = 0;
        bool accept = true;
        for (int i = 0; i < 4; i++) {
            x_err[i] = dt*(bhat1 * derivs[i] + bhat3 * k3[i] + bhat4 * k4[i] + bhat5 * k5[i] + bhat6 * k6[i] + bhat7 * k7[i]);
            x_err[i] = std::abs(x_err[i]) / (tol + tol*(std::abs(state[i]) + std::abs(derivs[i])));      
            // std::cout << std::abs(x_err[i]) << "\n";
            err = std::max(err, x_err[i]);
        }

        // std::cout << "err= " << err << "\n";

        // Compute new step size

        // std::cout << "intermediate val=" << 0.9*pow(err, -1.0/5.0) << "\n";
        double dt_new = dt*0.9*pow(err, -1.0/5.0);
        dt_new = std::max(dt_new, 0.2 * dt);  // Limit step size reduction
        dt_new = std::min(dt_new, 5.0 * dt);  // Limit step size increase
        if ((0.5 < err) & (err < 1.0)){
            dt_new = dt;
        }
        // dt_new = std::max(dt_new, 1e-9); // Limit smallest step size
        // std::cout << "dt_new= " << dt_new << "\t dt=" << dt << "\n";
        if (err <= 1.0) {
            // std::cout << "point accepted\n";
            // Accept the step
            t += dt;
            dt = std::min(dt_new, tmax - t);

            p.x = x_new[0];
            p.y = x_new[1];
            p.z = x_new[2];
            p.v_par = x_new[3];
        } else {
            // Reject the step and try again with smaller dt
            dt = dt_new;
        }

    }
    return;
}

// __global__ void particle_trace_kernel(particle_t* particles, double* workspaces, double* rrange_arr, double* zrange_arr, double* phirange_arr, double* quadpts_arr,
//                         double dt, double tmax, double m, double q, int nparticles){
//     int idx = threadIdx.x + blockIdx.x*blockDim.x;
//     if(idx < nparticles){
//         trace_particle(particles[idx], rrange_arr, zrange_arr, phirange_arr, quadpts_arr, dt, tmax, m, q);
//     }
// }

extern "C" vector<bool> gpu_tracing(py::array_t<double> quad_pts, py::array_t<double> rrange,
        py::array_t<double> phirange, py::array_t<double> zrange, py::array_t<double> xyz_init, double m, double q, double vtotal, py::array_t<double> vtang, 
        double tmax, double tol, bool vacuum, vector<double> phis, vector<shared_ptr<StoppingCriterion>> stopping_criteria, int nparticles){

    vector<vector<array<double, 5>>> res_all(nparticles);
    vector<vector<array<double, 6>>> res_phi_hits_all(nparticles);


    //  read data in from python
    auto ptr = xyz_init.data();
    int size = xyz_init.size();
    double xyz_init_arr[size];
    std::memcpy(xyz_init_arr, ptr, size * sizeof(double));

    // py::buffer_info xyz_buf = xyz_init.request();
    // double* xyz_init_arr = static_cast<double*>(xyz_buf.ptr);
    
    py::buffer_info vtang_buf = vtang.request();
    double* vtang_arr = static_cast<double*>(vtang_buf.ptr);

    // contsins b field and then curve distance
    py::buffer_info quadpts_buf = quad_pts.request();
    double* quadpts_arr = static_cast<double*>(quadpts_buf.ptr);

    py::buffer_info r_buf = rrange.request();
    double* rrange_arr = static_cast<double*>(r_buf.ptr);

    py::buffer_info phi_buf = phirange.request();
    double* phirange_arr = static_cast<double*>(phi_buf.ptr);

    py::buffer_info z_buf = zrange.request();
    double* zrange_arr = static_cast<double*>(z_buf.ptr);


    particle_t* particles =  new particle_t[nparticles];
    for(int i=0; i<nparticles; ++i){
        int start = 3*i;
        particles[i].x = xyz_init_arr[start];
        particles[i].y = xyz_init_arr[start+1];
        particles[i].z = xyz_init_arr[start+2];
        particles[i].v_par = vtang_arr[i];
        particles[i].v_perp = sqrt(vtotal*vtotal -  particles[i].v_par* particles[i].v_par);
        particles[i].has_left = false;
        
    }

    int workspace_size = 150;
    double* workspaces = new double[nparticles*workspace_size];

    // workspace index mapping 
    // 0-3 is the state x,y,z, v_par
    // 4-9 is the deriv at state = k1
    // 10-13 k_2 state
    // 14-19 k_2
    // 20-23 k3_state
    // 24-29 k3
    // 30-33 k4_state
    // 34-39 k4
    // 40-43 r_shape
    // 44-47 z_shape
    // 48-51 phi_shape
    // 52-55 r_dshape
    // 56-59 z_dshape
    // 60-63 phi_dshape
    // 64-67 B
    // 68-76 grad_B
    // 77-79 nabla_normB
    // 80-83 cross_prod
    // 84-87 k5_state
    // 88-93 k5
    // 94-97 k6_state
    // 98-103 k6
    // 104-107 k7_state
    // 108-113 k7

    
    // // std::cout << "particles initialized \n";

    double dt = 1e-4*0.5*M_PI/vtotal;
    for(int p=0; p<nparticles; ++p){
        // std::cout << "tracing particle " << p << "\n";
        trace_particle(particles[p], rrange_arr, zrange_arr, phirange_arr, quadpts_arr, dt, tmax, m, q);
    }

    /*
    particle_t* particles_d;
    hipMalloc((void**)&particles_d, nparticles * sizeof(particle_t));
    hipMemcpy(particles_d, particles, nparticles * sizeof(particle_t), hipMemcpyHostToDevice);

    double* rrange_d;
    hipMalloc((void**)&rrange_d, 3 * sizeof(double));
    hipMemcpy(rrange_d, rrange_arr, 3 * sizeof(double), hipMemcpyHostToDevice);

    double* zrange_d;
    hipMalloc((void**)&zrange_d, 3 * sizeof(double));
    hipMemcpy(zrange_d, zrange_arr, 3 * sizeof(double), hipMemcpyHostToDevice);

    double* phirange_d;
    hipMalloc((void**)&phirange_d, 3 * sizeof(double));
    hipMemcpy(phirange_d, phirange_arr, 3 * sizeof(double), hipMemcpyHostToDevice);


    double* quadpts_d;
    hipMalloc((void**)&quadpts_d, quad_pts.size() * sizeof(double));
    hipMemcpy(quadpts_d, quadpts_arr, quad_pts.size() * sizeof(double), hipMemcpyHostToDevice);

    double* workspaces_d;
    hipMalloc((void**)&workspaces_d, nparticles*workspace_size * sizeof(double));

    int nthreads = 1;
    int nblks = nparticles / nthreads + 1;
    particle_trace_kernel<<<nblks, nthreads>>>(particles_d, workspaces_d, rrange_d, zrange_d, phirange_d, quadpts_d, dt, tmax, m, q, nparticles);

    hipMemcpy(particles, particles_d, nparticles * sizeof(particle_t), hipMemcpyDeviceToHost);

    */
    vector<bool> particle_loss(nparticles);
    for(int i=0; i<nparticles; ++i){
        particle_loss[i] = particles[i].has_left;
    }

    delete[] workspaces;
    delete[] particles;

    return particle_loss;
}



