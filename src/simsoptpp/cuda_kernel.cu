#include "hip/hip_runtime.h"
// #include "simdhelpers.h" // import above cuda_runtime to prevent collision for rsqrt
#include <hip/hip_runtime.h>
#include <iostream>
#include "tracing.h"
#include <math.h>
#include "xtensor-python/pyarray.hpp"     // Numpy bindings
typedef xt::pyarray<double> PyArray;
#include "xtensor-python/pytensor.hpp"     // Numpy bindings
typedef xt::pytensor<double, 2, xt::layout_type::row_major> PyTensor;
using std::shared_ptr;
using std::vector;
namespace py = pybind11;

// #include <Eigen/Core>

#include "magneticfield.h"
#include "boozermagneticfield.h"
#include "regular_grid_interpolant_3d.h"

// #define dt 1e-7

// Particle Data Structure
typedef struct particle_t {
    double x;  // Position X
    double y;  // Position Y
    double z;  // Position Z
    double v_par; // Velocity parallel
    double v_perp; // Velocity perpendicular
    double dotx;
    double doty;
    double dotz;
    double dotv_par;
    bool has_left;
} particle_t;

__global__ void addKernel(int *c, const int* a, const int* b, int size){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx < size){
        c[idx] = a[idx] + b[idx];
    }
}

extern "C" void addKernelWrapper(int *c, const int *a, const int *b, int size){
    int *d_a, *d_b, *d_c;

    hipMalloc((void **)&d_a, size*sizeof(int));
    hipMalloc((void **)&d_b, size*sizeof(int));
    hipMalloc((void **)&d_c, size*sizeof(int));

    hipMemcpy(d_a, a, size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size*sizeof(int), hipMemcpyHostToDevice);

    addKernel<<<1, 256>>>(d_c, d_a, d_b, size);

    for(int i=0; i<size; ++i){
        // // std::cout << c[i] <<"\n";
    }

    hipMemcpy(c, d_c, size*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}



__global__ void particle_trace_kernel(MagneticField<xt::pytensor> field, const double* xyz_init_arr,
        double m, double q, double vtotal, const double* vtang_arr, double tmax, double tol, bool vacuum, int nparticles, 
        tuple<vector<array<double, 5>>, vector<array<double, 6>>>* out){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx < nparticles){

        int start = 3*idx;
        // array<double, 3> xyz_init_i = {xyz_init_arr[start], xyz_init_arr[start+1], xyz_init_arr[start+2]};
        // typename MagneticField<xt::pytensor>::Tensor2 xyz({{xyz_init_arr[start], xyz_init_arr[start+1], xyz_init_arr[start+2]}});
        // field.set_points(xyz);
        
    //     out[idx] = particle_guiding_center_tracing(field, xyz_init_i, m, q, vtotal, vtang_arr[idx], tmax, tol, vacuum, phis, stopping_criteria);
    //     // res_all[idx] = std::get<0>(out_i);
    //     // res_phi_hits_all[idx] = std::get<1>(out_i);
    }
}

void shape(double x, double* shape){
    shape[0] = (1.0-x)*(2.0-x)*(3.0-x)/6.0;
    shape[1] = x*(2.0-x)*(3.0-x)/2.0;
    shape[2] = x*(x-1.0)*(3.0-x)/2.0;
    shape[3] = x*(x-1.0)*(x-2.0)/6.0;
    return;         
}

void dshape(double x, double h, double* dshape){
    dshape[0] = (-(2.0-x)*(3.0-x)-(1.0-x)*(3.0-x)-(1.0-x)*(2.0-x))/(h*6.0);
    dshape[1] = ( (2.0-x)*(3.0-x)-x*(3.0-x)-x*(2.0-x))/(h*2.0);
    dshape[2] = ( (x-1.0)*(3.0-x)+x*(3.0-x)-x*(x-1.0))/(h*2.0);
    dshape[3] = ( (x-1.0)*(x-2.0)+x*(x-2.0)+x*(x-1.0))/(h*6.0);
    return;         
}

// state has 5 elements: current values of x,y,z,v_perp
// // compute derivs for: x, y, z, v_perp
// void calc_derivs(double* state, double* derivs, double* rrange_arr, double* zrange_arr, double* phirange_arr, double* quadpts_arr,
//                         double dt, double tmax, double m, double q) {
//     double x = state[0];
//     double y = state[1];
//     double z = state[2];
//     double v_perp = state[3];

//     double r_shape[4];
//     double phi_shape[4];
//     double z_shape[4];

//     double r_dshape[4];
//     double phi_dshape[4];
//     double z_dshape[4];

//     double B[3];
//     double grad_B[9];
//     double nabla_normB[3];
//     double cross_prod[3];

//     double r_grid_size = (rrange_arr[1] - rrange_arr[0]) / (rrange_arr[2]-1);
//     double phi_grid_size = 2*M_PI / phirange_arr[2];
//     double z_grid_size = (zrange_arr[1] - zrange_arr[0]) / (zrange_arr[2]-1);

//     // interpolate B field for current state
//     double r = sqrt(x*x + y*y);
//     double phi = atan2(y, x);

//     // index into mesh to obtain nearby points
//     int i = (int) ((r - rrange_arr[0]) / r_grid_size) + 1;
//     int j = (int) ((z - zrange_arr[0]) / z_grid_size) + 1;
//     int k = (int) (phi / phi_grid_size) + 1;



//     // normalized positions in local grid wrt e.g. r at index i
//     int nr = rrange_arr[2];
//     int nphi = phirange_arr[2];
//     int nz = zrange_arr[2];
//     double r_rel = (r -  (rrange_arr[0] + i*r_grid_size)) / r_grid_size;
//     double z_rel = (z -  (zrange_arr[0] + j*z_grid_size)) / z_grid_size;
//     double phi_rel = (phi - (k*phi_grid_size)) / phi_grid_size;


//     // std::cout << r << "\t" << -1*(r_rel*r_grid_size - r) << "\t" << r_grid_size << "\n";
//     // std::cout << z << "\t" << -1*(z_rel*z_grid_size - z) << "\t" << z_grid_size << "\n";
//     // std::cout << phi << "\t" << -1*(phi_rel*phi_grid_size - phi) << "\t" << phi_grid_size << "\n";
//     // std::cout << "using index " <<  (i*nz*nphi + j*nphi + k) << "\n";
//     // std::cout << quadpts_arr[4*(i*nz*nphi + j*nphi + k) + 3] << "\n";
//     // // std::cout << "grid point found \n";

//     // // std::cout << "r_rel " << r_rel << "\t" << z_rel << "\t" << phi_rel << "\n";

//     shape(r_rel, r_shape);
//     shape(z_rel, z_shape);
//     shape(phi_rel, phi_shape);


//     // // std::cout <<"shape set \n";
//     // accumulate interpolation of B
//     B[0] = 0.0;
//     B[1] = 0.0;            
//     B[2] = 0.0;

//     // interpolate the distance to the surface
//     double surface_dist = 0.0;

//     // // std::cout << "starting B accumulation\n";
//     // quad pts are indexed r z phi
//     bool is_lost = false;
//     for(int ii=0; ii<=3; ++ii){             
//         for(int jj=0; jj<=3; ++jj){                 
//             for(int kk=0; kk<=3; ++kk){
//                 int wrap_k = ((k+kk-1) % nphi) + 1;

//                 if ((i+ii >= 0 & i+ii < nr) & (j+jj >= 0 & j+jj < nz)){
//                     int start = 4*((i+ii)*nz*nphi + (j+jj)*nphi + (wrap_k));
//                     // // std::cout << "start=" << start << "\t" << 4*nr*nz*nphi << "\n";
//                     B[0] += quadpts_arr[start]   * r_shape[ii]*z_shape[jj]*phi_shape[kk];
//                     B[1] += quadpts_arr[start+1] * r_shape[ii]*z_shape[jj]*phi_shape[kk];
//                     B[2] += quadpts_arr[start+2] * r_shape[ii]*z_shape[jj]*phi_shape[kk];

//                     is_lost = is_lost || (quadpts_arr[start+3] < 0); 
//                     // // std::cout << ii << "\t" << jj << "\t" << kk << "\n";
//                     // // std::cout << "interp surface dist val: " << quadpts_arr[start+3] << "\n";
//                     surface_dist += quadpts_arr[start+3] * r_shape[ii]*z_shape[jj]*phi_shape[kk];
//                 } else{
//                     // // std::cout << "bad grid index for" << r << "\t" << phi << "\t" << z <<"\n"; 
//                 }

//             }
//         }
//     }

// }

// out contains derivatives for x , y, z, v_par, and then norm of B and surface distance interpolation
void calc_derivs(double* state, double* out, double* rrange_arr, double* zrange_arr, double* phirange_arr, double* quadpts_arr, double m, double q, double mu){
    double r_shape[4];
    double phi_shape[4];
    double z_shape[4];

    double r_dshape[4];
    double phi_dshape[4];
    double z_dshape[4];

    double B[3];
    double grad_B[9];
    double nabla_normB[3];
    double cross_prod[3];

    double r_grid_size = (rrange_arr[1] - rrange_arr[0]) / (rrange_arr[2]-1);
    double phi_grid_size = 2*M_PI / phirange_arr[2];
    double z_grid_size = (zrange_arr[1] - zrange_arr[0]) / (zrange_arr[2]-1);
    

    double x = state[0];
    double y = state[1];
    double z = state[2];
    double v_par = state[3];
    // double v_perp = state[4];

    // std::cout << "load v_par " << v_par;

    // magnetic field quad points are in cylindrical coordinates
    double r = sqrt(x*x + y*y);
    double phi = atan2(y, x); 
    
    // keep phi positive
    phi += (2*M_PI)*(phi < 0);
    
    // index into mesh to obtain nearby points
    // get correct "meta grid" for continuity
    int i = 4*((int) ((r - rrange_arr[0]) / r_grid_size) / 4);
    int j = 4*((int) ((z - zrange_arr[0]) / z_grid_size) / 4);
    int k = 4*((int) (phi / phi_grid_size) / 4);

    // std::cout << "indices: " << i << "\t" << r << "\t" << rrange_arr[0] << "\t" << r_grid_size << "\n";
    // std::cout << "position: " << x << "\t" << y << "\t" << z <<"\n";

    // normalized positions in local grid wrt e.g. r at index i
    int nr = rrange_arr[2];
    int nphi = phirange_arr[2];
    int nz = zrange_arr[2];
    double r_rel = (r -  (rrange_arr[0] + i*r_grid_size)) / r_grid_size;
    double z_rel = (z -  (zrange_arr[0] + j*z_grid_size)) / z_grid_size;
    double phi_rel = (phi - (k*phi_grid_size)) / phi_grid_size;


    // std::cout << r << "\t" << -1*(r_rel*r_grid_size - r) << "\t" << r_grid_size << "\n";
    // std::cout << z << "\t" << -1*(z_rel*z_grid_size - z) << "\t" << z_grid_size << "\n";
    // std::cout << phi << "\t" << -1*(phi_rel*phi_grid_size - phi) << "\t" << phi_grid_size << "\n";
    // std::cout << i << "\t" << j << "\t" << k << "\n";
    // std::cout << "using index " <<  (i*nz*nphi + j*nphi + k) << "\n";
    // std::cout << quadpts_arr[4*(i*nz*nphi + j*nphi + k) + 3] << "\n";
    // // std::cout << "grid point found \n";

    // // std::cout << "r_rel " << r_rel << "\t" << z_rel << "\t" << phi_rel << "\n";

    shape(r_rel, r_shape);
    shape(z_rel, z_shape);
    shape(phi_rel, phi_shape);


    // // std::cout <<"shape set \n";
    // accumulate interpolation of B
    B[0] = 0.0;
    B[1] = 0.0;            
    B[2] = 0.0;

    // interpolate the distance to the surface
    double surface_dist = 0.0;

    // // std::cout << "starting B accumulation\n";
    // quad pts are indexed r z phi
    bool is_lost = false;
    for(int ii=0; ii<=3; ++ii){             
        for(int jj=0; jj<=3; ++jj){                 
            for(int kk=0; kk<=3; ++kk){
                int wrap_k = ((k+kk) % nphi);
                if ((i+ii >= 0 & i+ii < nr) & (j+jj >= 0 & j+jj < nz)){
                    int start = 4*((i+ii)*nz*nphi + (j+jj)*nphi + (wrap_k));
                    // // std::cout << "start=" << start << "\t" << 4*nr*nz*nphi << "\n";
                    B[0] += quadpts_arr[start]   * r_shape[ii]*z_shape[jj]*phi_shape[kk];
                    B[1] += quadpts_arr[start+1] * r_shape[ii]*z_shape[jj]*phi_shape[kk];
                    B[2] += quadpts_arr[start+2] * r_shape[ii]*z_shape[jj]*phi_shape[kk];

                    is_lost = is_lost || (quadpts_arr[start+3] < 0); 
                    // // std::cout << ii << "\t" << jj << "\t" << kk << "\n";
                    // // std::cout << "interp surface dist val: " << quadpts_arr[start+3] << "\n";
                    surface_dist += quadpts_arr[start+3] * r_shape[ii]*z_shape[jj]*phi_shape[kk];
                } else{
                    // // std::cout << "bad grid index for" << r << "\t" << phi << "\t" << z <<"\n"; 
                }

            }
        }
    }

    // std::cout << "k " << k << "\t" << nphi << "\n";


    // std::cout << "is quad pt lost: " << is_lost << "\n";
    if(!is_lost){ // can't lose a particle if no quad pts are lost
        surface_dist = 1.0;    
    }
    // // std::cout << "B interpolated \n";

    // // std::cout << "r=" << r << "\t" << x << "\t" << y << "\t" << p.v_par << "\t" << surface_dist << "\n";

    // // std::cout << "particle not lost \n";

    //  Interpolate grad B: columns are partial deriv wrt r, z, phi, rows are entries of B
    //  row major order
    for(int ii=0; ii<9; ++ii){
        grad_B[ii] = 0.0;
    }
    dshape(r_rel, r_grid_size, r_dshape);
    dshape(phi_rel, phi_grid_size, phi_dshape);
    dshape(z_rel, z_grid_size, z_dshape);

    for(int ii=0; ii<=3; ++ii){             
        for(int jj=0; jj<=3; ++jj){                 
            for(int kk=0; kk<=3; ++kk){
                int wrap_k = ((k+kk) % nphi);
                if ((i+ii >= 0 & i+ii < nr) & (j+jj >= 0 & j+jj < nz)){
                    int start = 4*((i+ii)*nz*nphi + (j+jj)*nphi + (wrap_k));
                    // interpolate gradient for each entry of B, filling in each row of the gradient
                    for(int l=0; l<3; ++l){
                        double Bval = quadpts_arr[start+l];
                        grad_B[3*l]   += Bval * r_dshape[ii]*z_shape[jj]*phi_shape[kk];
                        grad_B[3*l+1] += Bval * r_shape[ii]*z_dshape[jj]*phi_shape[kk];
                        grad_B[3*l+2] += Bval * r_shape[ii]*z_shape[jj]*phi_dshape[kk];
                    }
                }

            }
        }
    }

    // // std::cout << "grad B interpolated \n";


    // convert gradient from cylindrical (r, z, phi) to cartesian coordinates (x, y, z)
    double c = cos(phi);
    double s = sin(phi);


    for(int l=0; l<3; ++l){ // iter over row
        double dfdr = grad_B[3*l];
        double dfdphi_divr = grad_B[3*l+2] / r;
        
        grad_B[3*l]   = c*dfdr - s*dfdphi_divr;
        grad_B[3*l+2] = grad_B[3*l+1]; // z index changes
        grad_B[3*l+1] = s*dfdr + c*dfdphi_divr;
    }

    // std::cout << "B" << B[0] << "\t" << B[1] << "\t" << B[2] << "\n";
    // std::cout << "grad_B" << grad_B[0] << "\t" << grad_B[1] << "\t" << grad_B[2] << "\n";
    // now compute derivatives

    // // std::cout << "starting updates \n";

    double normB = sqrt(B[0]*B[0] + B[1]*B[1] + B[2]*B[2]);


    // compute \nabla |B|
    //  \nabla |B| = (\nabla B  B) / (2 |B|)
    nabla_normB[0] = (grad_B[0]*B[0] + grad_B[1]*B[1] + grad_B[2]*B[2]) / (normB);
    nabla_normB[1] = (grad_B[3]*B[0] + grad_B[4]*B[1] + grad_B[5]*B[2]) / (normB);
    nabla_normB[2] = (grad_B[6]*B[0] + grad_B[7]*B[1] + grad_B[8]*B[2]) / (normB);

    // compute B \times \nabla |B|
    cross_prod[0] = B[1]*nabla_normB[2] - B[2]*nabla_normB[1];
    cross_prod[1] = B[2]*nabla_normB[0] - B[0]*nabla_normB[2];
    cross_prod[2] = B[0]*nabla_normB[1] - B[1]*nabla_normB[0];

    // std::cout << "compute x deriv: " << v_par << "\t" << B[0] << "\t" << normB << "\t" <<  v_par << "\t" << cross_prod[0] << "\t" << m << "\t" << q << "\n";

    double v_perp2 = 2*mu*normB;

    // std::cout << "should be 0: " << (0.5*v_perp2 + pow(v_par, 2))*cross_prod[0] * m/(q*pow(normB, 3)) << "\n";
    // std::cout << "should be positive " << v_par * B[0]/normB << "\n";
    // std::cout << "v_par" << v_par << "\n";

    out[0] = v_par * B[0]/normB + (0.5*v_perp2 + pow(v_par, 2))*cross_prod[0] * m/(q*pow(normB, 3));
    out[1] = v_par * B[1]/normB + (0.5*v_perp2 + pow(v_par, 2))*cross_prod[1] * m/(q*pow(normB, 3));
    out[2] = v_par * B[2]/normB + (0.5*v_perp2 + pow(v_par, 2))*cross_prod[2] * m/(q*pow(normB, 3));

    double BdotNablaNormB = B[0]*nabla_normB[0] + B[1]*nabla_normB[1] + B[2]*nabla_normB[2];
    out[3] = -mu*BdotNablaNormB/normB;
    out[4] = normB;
    out[5] = surface_dist;

}


void trace_particle(particle_t& p, double* rrange_arr, double* zrange_arr, double* phirange_arr, double* quadpts_arr,
                        double dt, double tmax, double m, double q){
    double mu;
    int nsteps = (int) (tmax / dt);
    double surface_dist;
    // // std::cout << tmax << "\t" << dt << "\t" << nsteps << "\n";
    // double r_shape[4];
    // double phi_shape[4];
    // double z_shape[4];

    // double r_dshape[4];
    // double phi_dshape[4];
    // double z_dshape[4];

    // double B[3];
    // double grad_B[9];
    // double nabla_normB[3];
    // double cross_prod[3];

    // double r_grid_size = (rrange_arr[1] - rrange_arr[0]) / (rrange_arr[2]-1);
    // double phi_grid_size = 2*M_PI / phirange_arr[2];
    // double z_grid_size = (zrange_arr[1] - zrange_arr[0]) / (zrange_arr[2]-1);
    

    double t = 0.0;
    // for(int time_step=0; time_step<nsteps; ++time_step){

    double state[4];
    state[0] = p.x;
    state[1] = p.y;
    state[2] = p.z;
    state[3] = p.v_par;
    // state[4] = p.v_perp;

    double derivs[5];

    // dummy call to get norm B
    calc_derivs(state, derivs, rrange_arr, zrange_arr, phirange_arr, quadpts_arr, m, q, -1);
    mu = p.v_perp*p.v_perp/(2*derivs[4]);

    double k2_state[4];
    double k3_state[4];
    double k4_state[4];

    // won't use the surface distance element
    double k2[5];
    double k3[5];
    double k4[5];

    
    while(t < tmax){
        std::cout << "position: " << p.x << "\t" << p.y << "\t" << p.z << "\t" << "t=" << t  << "\n";

        // std::cout << "Time: " << t << "\n";
        /*
        * Time step ODE
        * runge-kutta 4 (see https://lpsa.swarthmore.edu/NumInt/NumIntFourth.html)
        */

        // compute k1
        state[0] = p.x;
        state[1] = p.y;
        state[2] = p.z;
        state[3] = p.v_par;

        calc_derivs(state, derivs, rrange_arr, zrange_arr, phirange_arr, quadpts_arr, m, q, mu);

        // stop if particle lost
        surface_dist = derivs[5];
        if(surface_dist <= 0){
            std::cout << "particle lost: " << surface_dist << "\t" << t << "\t" << dt << "\n";
            p.has_left = true;
            return;
        }

        for(int i=0; i<4; ++i){
            k2_state[i] = state[i] + derivs[i]*dt/2;
        }
        calc_derivs(k2_state, k2, rrange_arr, zrange_arr, phirange_arr, quadpts_arr, m, q, mu);

        for(int i=0; i<4; ++i){
            k3_state[i] = state[i] + k2[i]*dt/2;
        }
        calc_derivs(k3_state, k3, rrange_arr, zrange_arr, phirange_arr, quadpts_arr, m, q, mu);

        for(int i=0; i<4; ++i){
            k4_state[i] = state[i] + k3[i]*dt;
        }
        calc_derivs(k4_state, k4, rrange_arr, zrange_arr, phirange_arr, quadpts_arr, m, q, mu);

        // update
        p.x +=     dt*(derivs[0] + 2*k2[0] + 2*k3[0] + k4[0])/6;
        p.y +=     dt*(derivs[1] + 2*k2[1] + 2*k3[1] + k4[1])/6;
        p.z +=     dt*(derivs[2] + 2*k2[2] + 2*k3[2] + k4[2])/6;
        p.v_par += dt*(derivs[3] + 2*k2[3] + 2*k3[3] + k4[3])/6;

        // // update
        // // // std::cout << "x update: " << p.x << "\t" <<  derivs[0] <<  "\t" << derivs[0] * dt << "\n";
        // p.x += derivs[0] * dt;
        // p.y += derivs[1] * dt;
        // p.z += derivs[2] * dt;
        // p.v_par += derivs[3] * dt;

        t += dt;
        // // std::cout << "updates complete \n";

    }
    return;
}



extern "C" vector<bool> gpu_tracing(py::array_t<double> quad_pts, py::array_t<double> rrange,
        py::array_t<double> phirange, py::array_t<double> zrange, py::array_t<double> xyz_init, double m, double q, double vtotal, py::array_t<double> vtang, 
        double tmax, double tol, bool vacuum, vector<double> phis, vector<shared_ptr<StoppingCriterion>> stopping_criteria, int nparticles){

    vector<vector<array<double, 5>>> res_all(nparticles);
    vector<vector<array<double, 6>>> res_phi_hits_all(nparticles);


    //  read data in from python
    py::buffer_info xyz_buf = xyz_init.request();
    double* xyz_init_arr = static_cast<double*>(xyz_buf.ptr);
    
    py::buffer_info vtang_buf = vtang.request();
    double* vtang_arr = static_cast<double*>(vtang_buf.ptr);

    // contsins b field and then curve distance
    py::buffer_info quadpts_buf = quad_pts.request();
    double* quadpts_arr = static_cast<double*>(quadpts_buf.ptr);

    py::buffer_info r_buf = rrange.request();
    double* rrange_arr = static_cast<double*>(r_buf.ptr);

    py::buffer_info phi_buf = phirange.request();
    double* phirange_arr = static_cast<double*>(phi_buf.ptr);

    py::buffer_info z_buf = zrange.request();
    double* zrange_arr = static_cast<double*>(z_buf.ptr);


    particle_t particles[nparticles];
    for(int i=0; i<nparticles; ++i){
        int start = 3*i;
        particles[i].x = xyz_init_arr[start];
        particles[i].y = xyz_init_arr[start+1];
        particles[i].z = xyz_init_arr[start+2];
        particles[i].v_par = vtang_arr[i];
        particles[i].v_perp = sqrt(vtotal*vtotal -  particles[i].v_par* particles[i].v_par);
        particles[i].has_left = false;
        
    }

    // // std::cout << "particles initialized \n";

    double dt = 1e-12;//  1e-4*0.5*M_PI/vtotal;
    for(int p=0; p<nparticles; ++p){
        // std::cout << "tracing particle " << p << "\n";
        trace_particle(particles[p], rrange_arr, zrange_arr, phirange_arr, quadpts_arr, dt, tmax, m, q);
    }

    vector<bool> particle_loss(nparticles);
    for(int i=0; i<nparticles; ++i){
        particle_loss[i] = particles[i].has_left;
    }
    return particle_loss;
}



